#include "hip/hip_runtime.h"
extern "C"
{
#include "sph/sph_blake.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"
#include "sph/sph_luffa.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_echo.h"
#include "sph/sph_fugue.h"
}

#include "miner.h"
#include "cuda_helper.h"

#include <stdio.h>
#include <memory.h>

//#define DR_TIMINGS

//#define DROP_OFFF

#ifdef DROP_OFFF
static uint16_t* d_poks[MAX_GPUS];
#endif

static uint32_t* d_hash[MAX_GPUS];
static uint64_t* d_roundInfo[MAX_GPUS];

extern "C" static void switch_hash(const void *input, void *output, int id)
{
	switch (id) {
	case 0:
		sph_keccak512_context ctx_keccak;
		sph_keccak512_init(&ctx_keccak);
		sph_keccak512(&ctx_keccak, input, 64);
		sph_keccak512_close(&ctx_keccak, output);
		break;
	case 1:
		sph_blake512_context ctx_blake;
		sph_blake512_init(&ctx_blake);
		sph_blake512(&ctx_blake, input, 64);
		sph_blake512_close(&ctx_blake, output);
		break;
	case 2:
		sph_groestl512_context ctx_groestl;
		sph_groestl512_init(&ctx_groestl);
		sph_groestl512(&ctx_groestl, input, 64);
		sph_groestl512_close(&ctx_groestl, output);
		break;
	case 3:
		sph_skein512_context ctx_skein;
		sph_skein512_init(&ctx_skein);
		sph_skein512(&ctx_skein, input, 64);
		sph_skein512_close(&ctx_skein, output);
		break;
	case 4:
		sph_luffa512_context ctx_luffa;
		sph_luffa512_init(&ctx_luffa);
		sph_luffa512(&ctx_luffa, input, 64);
		sph_luffa512_close(&ctx_luffa, output);
		break;
	case 5:
		sph_echo512_context ctx_echo;
		sph_echo512_init(&ctx_echo);
		sph_echo512(&ctx_echo, input, 64);
		sph_echo512_close(&ctx_echo, output);
		break;
	case 6:
		sph_shavite512_context ctx_shavite;
		sph_shavite512_init(&ctx_shavite);
		sph_shavite512(&ctx_shavite, input, 64);
		sph_shavite512_close(&ctx_shavite, output);
		break;
	case 7:
		sph_fugue512_context ctx_fugue;
		sph_fugue512_init(&ctx_fugue);
		sph_fugue512(&ctx_fugue, input, 64);
		sph_fugue512_close(&ctx_fugue, output);
		break;
	case 8:
		sph_simd512_context ctx_simd;
		sph_simd512_init(&ctx_simd);
		sph_simd512(&ctx_simd, input, 64);
		sph_simd512_close(&ctx_simd, output);
		break;
	case 9:
		sph_cubehash512_context ctx_cubehash;
		sph_cubehash512_init(&ctx_cubehash);
		sph_cubehash512(&ctx_cubehash, input, 64);
		sph_cubehash512_close(&ctx_cubehash, output);
		break;
	default:
		break;
	}
}

extern "C" static void shiftr_lp(const uint32_t *input, uint32_t *output, unsigned int shift)
{
	if (!shift) {
		memcpy(output, input, 64);
		return;
	}
	memset(output, 0, 64);
	for (int i = 0; i < 15; ++i) {
		output[i + 1] |= (input[i] >> (32 - shift));
		output[i] |= (input[i] << shift);
	}
	output[15] |= (input[15] << shift);
	return;
}

// CPU HASH
extern "C" void drophash(void *output, const void *input)
{
	sph_jh512_context ctx_jh;
	uchar _ALIGN(64) hash[2][64];
	uint32_t *phashA = (uint32_t *)hash[0];
	uint32_t *phashB = (uint32_t *)hash[1];

	sph_jh512_init(&ctx_jh);
	sph_jh512(&ctx_jh, (const void*)input, 80);
	sph_jh512_close(&ctx_jh, (void*)phashA);

	int startPosition = phashA[0] % 31;

	for (int i = startPosition; i < 31; i--) {
		int start = i % 10;
		for (int j = start; j < 10; j++) {
			shiftr_lp(phashA, phashB, (i & 3));
			switch_hash((const void*)phashB, (void*)phashA, j);
		}
		for (int j = 0; j < start; j++) {
			shiftr_lp(phashA, phashB, (i & 3));
			switch_hash((const void*)phashB, (void*)phashA, j);
		}
		i += 10;
	}
	for (int i = 0; i < startPosition; i--) {
		int start = i % 10;
		for (int j = start; j < 10; j++) {
			shiftr_lp(phashA, phashB, (i & 3));
			switch_hash((const void*)phashB, (void*)phashA, j);
		}
		for (int j = 0; j < start; j++) {
			shiftr_lp(phashA, phashB, (i & 3));
			switch_hash((const void*)phashB, (void*)phashA, j);
		}
		i += 10;
	}
	memcpy(output, phashA, 32);
}

// ------------------------------------------------------------------------------------------------

#ifdef DROP_OFFF
__global__ __launch_bounds__(128, 8)
void drop_get_poks_gpu(uint32_t threads, uint32_t *d_hash, uint16_t *d_poks)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		d_poks[thread] = ((uint16_t*)&d_hash[thread * 16U])[1];
	}
}

__host__
void drop_get_poks(int thr_id, uint32_t threads, uint32_t *d_hash, uint16_t* d_poks)
{
	const uint32_t threadsperblock = 128;
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	drop_get_poks_gpu << <grid, block >> > (threads, d_hash, d_poks);
}

#endif

extern void drop_jh512_cpu_init(int thr_id, uint32_t threads);
extern void drop_jh512_cpu_setBlock80(void *pdata);

extern void drop_jh512_cpu_hash_80a(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order,  uint64_t *d_roundInfo);
extern void drop_jh512_cpu_hash_80b(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order,  uint64_t *d_roundInfo);

extern void quark_keccak512_cpu_init(int thr_id, uint32_t threads);
extern void quark_keccak512_cpu_hash_64_drop(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order,  uint64_t *d_roundInfo, int round, int subRound);

extern void quark_blake512_cpu_init(int thr_id, uint32_t threads);
extern void quark_blake512_cpu_hash_64_drop(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_outputHash, int order,  uint64_t *d_roundInfo, int round, int subRound);

extern void quark_groestl512_cpu_init(int thr_id, uint32_t threads);
extern void quark_groestl512_cpu_hash_64_drop(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order,  uint64_t *d_roundInfo, int round, int subRound);

extern void quark_skein512_cpu_init(int thr_id, uint32_t threads);
extern void quark_skein512_cpu_hash_64_drop(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order,  uint64_t *d_roundInfo, int round, int subRound);

extern void x11_luffa512_cpu_init(int thr_id, uint32_t threads);
extern void x11_luffa512_cpu_hash_64_drop(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order,  uint64_t *d_roundInfo, int round, int subRound);

extern void x11_echo512_cpu_init(int thr_id, uint32_t threads);
extern void x11_echo512_cpu_hash_64_drop(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order,  uint64_t *d_roundInfo, int round, int subRound);

extern void x11_shavite512_cpu_init(int thr_id, uint32_t threads);
extern void x11_shavite512_cpu_hash_64_drop(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order,  uint64_t *d_roundInfo, int round, int subRound);

extern void x13_fugue512_cpu_init(int thr_id, uint32_t threads);
extern void x13_fugue512_cpu_hash_64_drop(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order,  uint64_t *d_roundInfo, int round, int subRound);

extern void drop_simd512_cpu_init(int thr_id, uint32_t threads);
extern void drop_simd512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order, uint64_t * d_roundInfo, int round, int subRound);

extern void x11_cubehash512_cpu_init(int thr_id, uint32_t threads);
extern void x11_cubehash512_cpu_hash_64_drop(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order,  uint64_t *d_roundInfo, int round, int subRound);


extern "C" void drophash_pok(void *output, uint32_t *pdata, bool xnonce)
{
	const uint32_t version = pdata[0] & 0x0000FFFF;
	uint32_t _ALIGN(64) hash[8];
	uint32_t pok;

	pdata[0] = version;
	drophash(hash, pdata);

	// fill PoK
	pok = version | (hash[0] & 0xFFFF0000);
	if (pdata[0] != pok) {
		pdata[0] = pok;
		drophash(hash, pdata);
	}
	if (xnonce)	pdata[22] = pok;
	memcpy(output, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_drop(int thr_id, uint32_t *pdata, const uint32_t *ptarget,
	uint32_t max_nonce, unsigned long *hashes_done)
{
#ifdef DROP_OFFF
	uint32_t _ALIGN(64) tmpdata[20];
#else
	const uint32_t oP = pdata[22];
#endif
	const uint32_t first_nonce = pdata[19];

	uint32_t throughpt = device_intensity(thr_id, __func__, 1U << 19);
	const uint32_t throughput = min(throughpt, max_nonce - first_nonce);

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x0000ff;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		//hipDeviceReset();
		//hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

		hipMalloc(&d_hash[thr_id], throughput * 16 * sizeof(uint32_t));
		hipMalloc(&d_roundInfo[thr_id], throughput * sizeof(uint64_t));
#ifdef DROP_OFFF
		hipMalloc(&d_poks[thr_id], throughput * sizeof(uint16_t));
#endif
		CUDA_SAFE_CALL(hipGetLastError());

		drop_jh512_cpu_init(thr_id, throughput);
		quark_keccak512_cpu_init(thr_id, throughput);
		quark_blake512_cpu_init(thr_id, throughput);
		quark_groestl512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);
		x11_luffa512_cpu_init(thr_id, throughput);
		x11_echo512_cpu_init(thr_id, throughput);
		x11_shavite512_cpu_init(thr_id, throughput);
		x13_fugue512_cpu_init(thr_id, throughput);
		drop_simd512_cpu_init(thr_id, throughput);
		x11_cubehash512_cpu_init(thr_id, throughput);

		cuda_check_cpu_init(thr_id, throughput);
		init[thr_id] = true;
	}

	drop_jh512_cpu_setBlock80((void*)pdata);
	cuda_check_cpu_setTarget(ptarget);

#ifdef DR_TIMINGS
	UINT timePeriod = 1;
	timeBeginPeriod(timePeriod);
	long prTime;
#endif
	do {
#ifdef DR_TIMINGS
		prTime = timeGetTime();
#endif
		int order = 0;
		// Hash with CUDA - round 1 of 2
		drop_jh512_cpu_hash_80a(thr_id, throughput, pdata[19], d_hash[thr_id], order++,d_roundInfo[thr_id]);
		//hipDeviceSynchronize();

		for (int i = 0; i < 5; i++) {
			for (int j = 0; j < 10; j++) {
				quark_keccak512_cpu_hash_64_drop(thr_id, throughput, pdata[19], d_hash[thr_id], order++, d_roundInfo[thr_id], i, j);
				quark_blake512_cpu_hash_64_drop(thr_id, throughput, pdata[19], d_hash[thr_id], order++, d_roundInfo[thr_id], i, j);
				quark_groestl512_cpu_hash_64_drop(thr_id, throughput, pdata[19], d_hash[thr_id], order++, d_roundInfo[thr_id], i, j);
				quark_skein512_cpu_hash_64_drop(thr_id, throughput, pdata[19], d_hash[thr_id], order++, d_roundInfo[thr_id], i, j);
				x11_luffa512_cpu_hash_64_drop(thr_id, throughput, pdata[19], d_hash[thr_id], order++, d_roundInfo[thr_id], i, j);
				x11_echo512_cpu_hash_64_drop(thr_id, throughput, pdata[19], d_hash[thr_id], order++, d_roundInfo[thr_id], i, j);
				x11_shavite512_cpu_hash_64_drop(thr_id, throughput, pdata[19], d_hash[thr_id], order++, d_roundInfo[thr_id], i, j);
				x13_fugue512_cpu_hash_64_drop(thr_id, throughput, pdata[19], d_hash[thr_id], order++, d_roundInfo[thr_id], i, j);
				drop_simd512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], order++, d_roundInfo[thr_id], i, j);
				x11_cubehash512_cpu_hash_64_drop(thr_id, throughput, pdata[19], d_hash[thr_id], order++, d_roundInfo[thr_id], i, j);
				//hipDeviceSynchronize();
			}
			//applog(LOG_DEBUG, "%d", i); CUDA_SAFE_CALL(hipGetLastError());
		}
#ifdef DROP_OFFF
		drop_get_poks(thr_id, throughput, d_hash[thr_id], d_poks[thr_id]);
#endif
		// Hash with CUDA - round 2 of 2
		drop_jh512_cpu_hash_80b(thr_id, throughput, pdata[19], d_hash[thr_id], order++, d_roundInfo[thr_id]);
		//hipDeviceSynchronize();

		for (int i = 0; i < 5; i++) {
			for (int j = 0; j < 10; j++) {
				quark_keccak512_cpu_hash_64_drop(thr_id, throughput, pdata[19], d_hash[thr_id], order++, d_roundInfo[thr_id], i, j);
				quark_blake512_cpu_hash_64_drop(thr_id, throughput, pdata[19], d_hash[thr_id], order++, d_roundInfo[thr_id], i, j);
				quark_groestl512_cpu_hash_64_drop(thr_id, throughput, pdata[19], d_hash[thr_id], order++, d_roundInfo[thr_id], i, j);
				quark_skein512_cpu_hash_64_drop(thr_id, throughput, pdata[19], d_hash[thr_id], order++, d_roundInfo[thr_id], i, j);
				x11_luffa512_cpu_hash_64_drop(thr_id, throughput, pdata[19], d_hash[thr_id], order++, d_roundInfo[thr_id], i, j);
				x11_echo512_cpu_hash_64_drop(thr_id, throughput, pdata[19], d_hash[thr_id], order++, d_roundInfo[thr_id], i, j);
				x11_shavite512_cpu_hash_64_drop(thr_id, throughput, pdata[19], d_hash[thr_id], order++, d_roundInfo[thr_id], i, j);
				x13_fugue512_cpu_hash_64_drop(thr_id, throughput, pdata[19], d_hash[thr_id], order++, d_roundInfo[thr_id], i, j);
				drop_simd512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], order++, d_roundInfo[thr_id], i, j);
				x11_cubehash512_cpu_hash_64_drop(thr_id, throughput, pdata[19], d_hash[thr_id], order++,d_roundInfo[thr_id], i, j);
				//hipDeviceSynchronize();
			}
		}

#ifdef DR_TIMINGS
		prTime = timeGetTime() - prTime;
		printf("\nTotal time elapsed: %f seconds\n", (prTime / 1000.0));
		timeEndPeriod(timePeriod);
#endif

		uint32_t foundNonce = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
		if (foundNonce != UINT32_MAX)
		{
			uint32_t vhash64[8];
			uint32_t oldp0 = pdata[0];
			uint32_t oldp19 = pdata[19];
#ifdef DROP_OFFF
			const uint32_t version = pdata[0] & 0x0000FFFF;
			memcpy(tmpdata, pdata, 80);

			uint32_t offset = foundNonce - pdata[19];
			uint32_t pok = 0;
			uint16_t h_pok;

			*hashes_done = pdata[19] - first_nonce + throughput;

			hipMemcpy(&h_pok, d_poks[thr_id] + offset, sizeof(uint16_t), hipMemcpyDeviceToHost);
			pok = version | (0x10000UL * h_pok);
			pdata[0] = pok; pdata[19] = foundNonce;
			drophash(vhash64, pdata);
			if (vhash64[7] <= ptarget[7] && fulltest(vhash64, ptarget)) {
				int res = 1;
				uint32_t secNonce = cuda_check_hash_suppl(thr_id, throughput, oldp19, d_hash[thr_id], 1);
				if (secNonce != 0) {
					offset = secNonce - oldp19;
					hipMemcpy(&h_pok, d_poks[thr_id] + offset, sizeof(uint16_t), hipMemcpyDeviceToHost);
					pok = version | (0x10000UL * h_pok);
					memcpy(tmpdata, pdata, 80);
					tmpdata[0] = pok; tmpdata[19] = secNonce;
					drophash(vhash64, tmpdata);
					if (vhash64[7] <= ptarget[7] && fulltest(vhash64, ptarget)) {
						pdata[21] = secNonce;
						pdata[22] = pok;
						res++;
					}
				}
#else

			const uint32_t Htarg = ptarget[7];

			*hashes_done = pdata[19] - first_nonce + throughput;

			pdata[19] = foundNonce;
			drophash_pok(vhash64, pdata, false);

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				uint32_t secNonce = cuda_check_hash_suppl(thr_id, throughput, oldp19, d_hash[thr_id], 1);
				if (secNonce != 0)
				{
					pdata[19] = secNonce;
					drophash_pok(vhash64, pdata, true);
					pdata[19] = foundNonce;
					if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
					{
						pdata[21] = secNonce;
						res++;
					}
					else
						pdata[22] = oP;
				}
#endif
				return res;
			}
			else {
				applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", device_map[thr_id], foundNonce);

				pdata[19]++;
				pdata[0] = oldp0;
			}
		}
		else
			pdata[19] += throughput;

	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce + 1;
	return 0;
}
